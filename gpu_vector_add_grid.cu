
#include <hip/hip_runtime.h>
#include <iostream>

const int N = 65537;

__global__ void add(int *a, int *b, int *c)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    while (tid < N)
    {
        c[tid] = a[tid] + b[tid];
        tid += blockDim.x * gridDim.x;
    }
}

int main(void)
{
   int a[N], b[N], c[N];
   int *dev_a, *dev_b, *dev_c;
   for (int i = 0; i < N; ++i)
   {
       a[i] = i;
       b[i] = i;
   }
   hipMalloc((void**)&dev_a, N*sizeof(int));
   hipMalloc((void**)&dev_b, N*sizeof(int));
   hipMalloc((void**)&dev_c, N*sizeof(int));
   
   hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
   hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

   add<<<128, 128>>>(dev_a, dev_b, dev_c);

   hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

   for (int i=0; i < N; ++i)
   {
       std::cout << a[i] << "+" << b[i] << "=" << c[i] << std::endl;

   }
   return 0;
}
