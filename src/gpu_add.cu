﻿
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void add(int a, int *b, int *c) //声明需要在GPU中执行的核函数
{
	*c = a + *b;
}

int main(void)
{
	int b, c;
	b = 3;
	int *dev_b, *dev_c; //声明GPU需要使用到的指针
	hipMalloc((void **)&dev_b, sizeof(int)); //在GPU内存中为dev_b分配内存
	hipMalloc((void **)&dev_c, sizeof(int)); //在GPU内存中为dev_c分配内存
	hipMemcpy(dev_b, &b, sizeof(int), hipMemcpyHostToDevice); //将核函数需要的数据复制到设备内存
	add<<<1,1>>>(2, dev_b, dev_c); //调用核函数
	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost); //将计算结果dev_c中的数据从设备内存复制到主机内存
	std::cout << 2 << " + " << b << " = " << c << std::endl;
	hipFree(dev_b); //释放设备内存
	hipFree(dev_c);
	return 0;
}
